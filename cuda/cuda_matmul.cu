#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <pybind11/pybind11.h>
#include <iostream>
#include <iomanip>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace py = pybind11;

/*
*********************************************************************
function name: matmul_cuda
description: dot product of two arbitrarily sized matrices.
parameters:
  image: Input image of size m X n.
  weight: weight kernel of size n X k.
  bias: bias per output channel.
  output: output image of size m x k.
  m,n,k: sizes of matrices.
  batch_size: Number of images in each batch.
return: none
Acknowledgement: Original code from 'lzhengchun/matrix-cuda' on github.
link: https://github.com/lzhengchun/matrix-cuda/blob/master/matrix_cuda.cu
*********************************************************************
*/
__global__ void matmul_cuda(
  const float *image,
  const float *weight,
	const float *bias,
  float *output,
  const int m,
  const int n,
  const int k,
  const int batch_size)
{

    // This code doesn't really get much faster using shared memory, since
    // accesses to the image matrix are all sequential anyway. The first access
    // already caches everything, making shared memory useless.

    int img = blockIdx.z * blockDim.z + threadIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f, product_appx = 0.0f, product = 0.0f;
    if( col < k && row < m && img < batch_size){
        for(int i = 0; i < n; i++){
						sum += image[(img*m*n)+(row*n) + i] * weight[i * k + col];
				}
        output[(img*m*k)+(row*k) + col] = sum + bias[col];
    }
}

__global__ void my_matmul_cuda(
  const float **image,
  const float *weight,
	const float *bias,
  float *output,
  const int l,
  const int m,
  const int n,
  const int *pixel_counts,
  const int *feature_counts,
  const int k,
  const int batch_size)
{

  //CONVIENE CHE OGNI BLOCCO DI FEATURE SIA VISTO IN DUE DIMENSIONI OSSIA #PIXEL(BATCHxWIDTHxHEIGHT) E #FEATURES. ENTRAMBI QUESTI DUE VALORI CAMBIANO PER OGNI BLOCCO E VA
  //TENUTA TRACCIA DELLE DIMENSIONI DI OGNI BLOCCO IN DUE VETTORI pixel_counts E feature_counts

    // This code doesn't really get much faster using shared memory, since
    // accesses to the image matrix are all sequential anyway. The first access
    // already caches everything, making shared memory useless.
    //int img = blockIdx.z * blockDim.z + threadIdx.z;
    int pixel = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f, product_appx = 0.0f, product = 0.0f;
    if(col < k && pixel < m){
      for(int i=0; i<l; i++){
        for(int j = 0; j < feature_counts[i]; j++){
          sum += image[i][((pixel*pixel_counts[i]/m) * pixel_counts[i] * feature_counts[i]) + j] * weight[j * k + col];
        }
      }
      output[(pixel*m*k) + col] = sum + bias[col];
    }

    return;


    /*int img = 0;//blockIdx.z * blockDim.z + threadIdx.z;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f, product_appx = 0.0f, product = 0.0f;
    if( col < k && row < m && img < batch_size){
        for(int i = 0; i < n; i++){
						sum += image[(img*m*n)+(row*n) + i] * weight[i * k + col];
				}
        output[(img*m*k)+(row*k) + col] = sum + bias[col];
    }*/
}

/*
*********************************************************************
function name: conv_forward
description: convolutional layer that calls the matmul cuda kernel.
parameters:
  image: Input image of size m X n.
  weight: weight kernel of size n X k.
  bias: bias per output channel.
  m,n,k: sizes of matrices.
  b: Number of images in each batch.
return:
  output: output image of size m x k.
*********************************************************************
*/
torch::Tensor conv_forward(
  torch::Tensor input,
  torch::Tensor weight,
  torch::Tensor bias,
	int m,
	int n,
	int k,
	int b
) {

  // Create an output of size b X m X k, directly on the GPU.
	auto options = torch::TensorOptions().device(torch::kCUDA, 0);
	auto output = torch::zeros({b, m, k}, options);

  // Use this block size to not exceed 1024 threads across all 3 dimensions.
  // You can also do dimblock(16 x 16 x 4) to use all 1024 threads if your
  // batches are small.
	unsigned int block_size = 8;
	unsigned int grid_rows = (m + block_size - 1) / block_size;
	unsigned int grid_cols = (k + block_size - 1) / block_size;
	unsigned int grid_images = (b + block_size - 1) / block_size;

	dim3 dimGrid(grid_cols, grid_rows, grid_images);
	dim3 dimBlock(block_size, block_size, block_size);

  // This is not the 'pytorch recommended way' of launching this kernel.
  // But it works just fine so I've left it this way since it is easier to debug
  // if there is an issue launching the kernel for example.

	matmul_cuda<<<dimGrid, dimBlock>>>(
		input.data_ptr<float>(),
		weight.data_ptr<float>(),
		bias.data_ptr<float>(),
		output.data_ptr<float>(),
		m, n, k, b
	);

  hipDeviceSynchronize();
  return output;
}

/*
*********************************************************************
function name: linear_forward
description: linear layer that calls the matmul cuda kernel.
parameters:
  image: Input image of size m X n.
  weight: weight kernel of size n X k.
  bias: bias per output channel.
  m,n,k: sizes of matrices.
return:
  output: output image of size m x k.
*********************************************************************
*/
torch::Tensor old_linear_forward(
  torch::Tensor input,
  torch::Tensor weight,
  torch::Tensor bias,
	int m,
	int n,
	int k
) {

	auto options = torch::TensorOptions().device(torch::kCUDA, 0);
	auto output = torch::zeros({m,k}, options);

	unsigned int block_size = 32;
	unsigned int grid_rows = (m + block_size - 1) / block_size;
	unsigned int grid_cols = (k + block_size - 1) / block_size;

	dim3 dimGrid(grid_cols, grid_rows);
	dim3 dimBlock(block_size, block_size);

  // Linear layers have a vector input. But to re-use the matmul kernel,
  // just pass in a 'batch' of inputs as an m X n matrix, to be multiplied
  // by the n x k weights, to get 'm' output images.

	matmul_cuda<<<dimGrid, dimBlock>>>(
		input.data_ptr<float>(),
		weight.data_ptr<float>(),
		bias.data_ptr<float>(),
		output.data_ptr<float>(),
		m, n, k, 1 // Pass in b=1 since there is no z-dimension for linear layers
	);

  hipDeviceSynchronize();
  return output;
}

//My new linear forward!

torch::Tensor linear_forward(
  std::vector<torch::Tensor> inputs,
  torch::Tensor weight,
  torch::Tensor bias,
	//int m,//batch size
	//int n,//number of features
	int k//output shape
) {

  int l = inputs.size();//number of input tensors
  int m = 0;//pixel
  int n = 0;//feature
  //int w = 0;//width or height of feature maps
  
  std::vector<int> pixel_counts;//number of pixels per layer
  pixel_counts.reserve(l);
  std::vector<int> feature_counts;//number of features per layer
  feature_counts.reserve(l);
  std::vector<const float*> input_ptrs;
  input_ptrs.reserve(l);
  for (const auto& input : inputs) {
    m = std::max(m, int(input.size(0)));//input.size(0);
    //w = std::max(w, int(input.size(1)));
    n += input.size(1);

    pixel_counts.push_back(input.size(0));
    feature_counts.push_back(input.size(1));
    input_ptrs.push_back(input.data_ptr<float>());
  }

	auto options = torch::TensorOptions().device(torch::kCUDA, 0);
	auto output = torch::zeros({m, k}, options);

	unsigned int block_size = 32;
	unsigned int grid_pixels = (m + block_size - 1) / block_size;
	unsigned int grid_cols = (k + block_size - 1) / block_size;

	dim3 dimGrid(grid_cols, grid_pixels);
	dim3 dimBlock(block_size, block_size);

  // Linear layers have a vector input. But to re-use the matmul kernel,
  // just pass in a 'batch' of inputs as an m X n matrix, to be multiplied
  // by the n x k weights, to get 'm' output images.

	my_matmul_cuda<<<dimGrid, dimBlock>>>(
		input_ptrs.data(),//input.data_ptr<float>(),
		weight.data_ptr<float>(),
		bias.data_ptr<float>(),
		output.data_ptr<float>(),
		l, m, n, pixel_counts.data(), feature_counts.data(), k, 1 // Pass in b=1 since there is no z-dimension for linear layers
	);

  hipDeviceSynchronize();
  return output;
}

// Binding to generate the .so file, to call from python.
PYBIND11_MODULE(cuda_layers, m) {
  m.doc() = "Implementation of forward pass of conv and linear layers in CUDA";
  m.def("conv_forward", &conv_forward, "conv_forward (CUDA)");
	m.def("linear_forward", &linear_forward, "linear_forward (CUDA)");
}
